/*
 * "sample.cu"
 *
 * An example for CUDA: summing vectors
 * 		c[i] = a[i]+b[i], where a[i]=i, b[i]=2*i
 *
 * This program implements the case above on both
 * CPU and GPU, check their results and also
 * compares their performances.
 *
 * Prefixes:
 * 		"d_" indicates device (GPU) memory pointer;
 *		"h_" indicates host (CPU) memory pointer.
 *
 * Same timing function is called in both cases.
 */

#include <stdio.h>
#include <stdlib.h>
//includes CUDA
#include "hip/hip_runtime.h"



#define	N	(1000*1000)
#define	block_size	512// no more than 512
#define block_num	(N+block_size-1)/block_size// no more than 65535

// return type of a CUDA kernel should always be "void"
__global__ void kernel_sum(int *d_a, int *d_b, int *d_c, int num){
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	if(tid<num){
		d_a[tid] = tid;
		d_b[tid] = tid*2;
		d_c[tid] = d_a[tid]+d_b[tid];
	}
}

double gpusum(int *result){
	// create cudaEvents for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory on device (GPU)
	int *d_a, *d_b, *d_c;
	hipMalloc((void **)&d_a, N*sizeof(int));
	hipMalloc((void **)&d_b, N*sizeof(int));
	hipMalloc((void **)&d_c, N*sizeof(int));

	hipEventRecord(start, 0);// record start

	// put everything you want to record here
	kernel_sum<<<block_num, block_size>>>(d_a, d_b, d_c, N);// launch the kernel

	hipEventRecord(stop, 0);// record stop
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);// calculate elapsedTime
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// copy data from device (GPU) to host (CPU)
	hipMemcpy(result, d_c, N*sizeof(int), hipMemcpyDeviceToHost);
	return double(elapsedTime);
}

double cpusum(int *result){
	int *h_a = new int[N];
	int *h_b = new int[N];

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);// record start

	for(int i=0; i<N; i++){
		h_a[i] = i;
		h_b[i] = i*2;
		result[i] = h_a[i]+h_b[i];
	}

	hipEventRecord(stop, 0);// record stop
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);// calculate elapsedTime
	hipEventDestroy(start);
	hipEventDestroy(stop);

	delete [] h_a;
	delete [] h_b;

	return double(elapsedTime);
}

bool check_result(int *vec_x, int *vec_y, int num){
	for(int i=0; i<num; i++)
		if(vec_x[i] != vec_y[i])
			return false;
	return true;
}

int main(){
	int *result_cpu = new int[N];
	int *result_gpu = new int[N];

	double time_cpu = cpusum(result_cpu);
	double time_gpu = gpusum(result_gpu);

	if(check_result(result_cpu, result_gpu, N)){
		printf("Results are correct!\n");
		printf("CPU Time:\t%f ms.\n", time_cpu);
		printf("GPU Time:\t%f ms.\n", time_gpu);
		printf("Speedup:\t%f.\n", time_cpu/time_gpu);
	}
	else{
		printf("Results are incorrect!\n");
	}

	delete [] result_cpu;
	delete [] result_gpu;
	return 0;
}
